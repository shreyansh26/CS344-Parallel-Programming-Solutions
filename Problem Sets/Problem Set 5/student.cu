#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  if(idx >= numVals)
      return;

  atomicAdd(&histo[vals[idx]], 1);
}

int get_max_size(int n, int dim) {
  return (int)ceil((float) n / (float) dim) + 1;
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free
  dim3 thread_dim(numBins);
  dim3 block_dim(get_max_size(numElems, numBins));

  yourHisto<<<block_dim, thread_dim>>>(d_vals, d_histo, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
